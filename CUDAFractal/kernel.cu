#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include "complex.h"
#include "window.h";

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ const double xMin = -2;
__device__ const double xMax = 2;
__device__ const double yMin = -2;
__device__ const double yMax = 2;
__device__ const double zoom = 1;

__global__ void iteration(int maxIt, int length, int width, int height, int *vec, bool *abort)
{
	int loop;
	double c;
	double imZ;
	double rlZ;
	double imC;
	double rlC;
	double xzoom = ((xMax - xMin) / width);
	double yzoom = ((yMax - yMin) / height);

	for (int i = threadIdx.x; i < length; i += blockDim.x + blockIdx.x)
	{
		if (*abort)
			break;

		rlC = xzoom * (i / height) - abs(xMin);
		imC = yzoom * (i % height) - abs(yMin);

		c = 0;
		imZ = 0;
		rlZ = 0;
		loop = 0;

		while (loop < maxIt && c <= 4)
		{
			loop++;

			Pow(&imZ, &rlZ);
			Add(&imZ, &rlZ, imC, rlC);
			Norm(&c, imZ, rlZ);
		}

		vec[i] = loop;
	}
}

int main()
{
	hipFree(0);

	int s;
	cout << "Size: ";
	cin >> s;
	cout << endl;

	Vector2 size(s, s);

	int N = (int)(size.X * size.Y * 4);
	int it = 2000;

	int *vec = new int[N];
	hipMallocManaged(&vec, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		vec[i] = 200;
	}

	bool *habort = new bool;
	bool *dabort = new bool;
	hipHostAlloc(&habort, sizeof(bool), hipHostMallocDefault);
	hipMalloc(&dabort, sizeof(bool));

	hipStream_t stream;
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

	*habort = false;
	hipMemcpyAsync(dabort, habort, sizeof(bool), hipMemcpyHostToDevice, stream);

	iteration<<<10, 1024, 0, stream>>>(it, N, size.X * 2, size.Y * 2, vec, dabort);

	//Sleep(1000);

	//hipMemcpy(habort, new bool(true), sizeof(bool), hipMemcpyHostToHost);
	//gpuErrchk(hipMemcpyAsync(dabort, habort, sizeof(bool), hipMemcpyHostToDevice, stream));

	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipPeekAtLastError());

	createWindow(size, N, it, vec);

	hipStreamDestroy(stream);
	hipFree(vec);
	hipFree(dabort);

	return 0;
}