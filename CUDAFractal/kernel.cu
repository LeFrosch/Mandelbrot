#include "hip/hip_runtime.h"
#define ADJUST 1

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include "complex.h"
#include "window.h";
#include "parser.h"

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ const double xMin = -2;
__device__ const double xMax = 2;
__device__ const double yMin = -2;
__device__ const double yMax = 2;
__device__ const double zoom = 1;

__global__ void iteration(int maxIt, int length, int width, int height, int *vec)
{
	int loop;
	double c;
	double imZ;
	double rlZ;
	double imC;
	double rlC;
	double xzoom = ((xMax - xMin) / width);
	double yzoom = ((yMax - yMin) / height);

	for (int i = threadIdx.x; i < length; i += blockDim.x + blockIdx.x)
	{
		rlC = xzoom * (i / height) - abs(xMin);
		imC = yzoom * (i % height) - abs(yMin);

		c = 0;
		imZ = 0;
		rlZ = 0;
		loop = 0;

		while (loop < maxIt && c <= 4)
		{
			loop++;

			Pow(&imZ, &rlZ);
			Add(&imZ, &rlZ, imC, rlC);
			Norm(&c, imZ, rlZ);
		}

		vec[i] = loop;
	}
}

int main(int argc, char *argv[])
{
	parse(argc, argv);

	hipFree(0);

	int s;
	cout << "Size: ";
	cin >> s;
	cout << endl;

	Vector2 size(s, s);

	int N = (int)(size.X * size.Y * 4);
	int it = 2000;

	int *hvec = (int*)malloc(N * sizeof(int));
	for (int i = 0; i < N; i++)
		hvec[i] = 200;

	int *dvec = 0;
	hipMalloc((void**)&dvec, N * sizeof(int));
	hipMemcpy(dvec, hvec, N * sizeof(int), hipMemcpyHostToDevice);


	iteration<<<10, 1024>>>(it, N, size.X * 2, size.Y * 2, dvec);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipPeekAtLastError());

	hipMemcpy(hvec, dvec, N * sizeof(int), hipMemcpyDeviceToHost);

	createWindow(size, N, it, hvec);

	hipFree(hvec);
	hipFree(dvec);

	return 0;
}